#include "detect.h"

int detectBestGPU(){
     hipDeviceProp_t prop;
     //hipError_t ret;
     int nb_devices=1;
     int best=0;
     int max_count=0;
     hipGetDeviceCount((&nb_devices));
     for (int i =0; i<nb_devices;i++){
	hipGetDeviceProperties ( (&prop), i );
	if (prop.multiProcessorCount>max_count){
		best=i;
		max_count=prop.multiProcessorCount;
        }
     }
     return best;
}

hipError_t setBestGPU()
{
    int device = detectBestGPU();
    hipDeviceProp_t prop;
    hipGetDeviceProperties ( (&prop), device );
    hipError_t ret = hipSetDevice ( device );
    printf("%s has been choosed\n",prop.name); 
    return ret;
}



