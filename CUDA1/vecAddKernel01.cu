
#include <hip/hip_runtime.h>


__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
	int blockStartIndex  = blockIdx.x * blockDim.x * N;
    int threadStartIndex = blockStartIndex + (threadIdx.x * N);
    int threadEndIndex   = threadStartIndex + N;
    int i;
    int add;
    printf("hello\n");
    for( i=threadStartIndex; i<threadEndIndex; i++ ){
    	add = ((i - threadStartIndex)*blockDim.x) + threadStartIndex;
		C[add] = A[add] + B[add];
    }
}
