
#include <hip/hip_runtime.h>


__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
    int blockStartIndex  = blockIdx.x * blockDim.x * N;
    int threadStartIndex = blockStartIndex + (threadIdx.x * N);
    int threadEndIndex   = threadStartIndex + N;
    int i;

    for( i=threadStartIndex; i<threadEndIndex; ++i ){
		C[threadStartIndex + (blockDim.x * i)] = A[threadStartIndex + (blockDim.x * i)] + B[threadStartIndex + (blockDim.x * i)];
    }
}
