
#include <hip/hip_runtime.h>


__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
    int blockStartIndex  = blockIdx.x * blockDim.x * N;
    int threadStartIndex = blockStartIndex + (threadIdx.x * N);
    int threadEndIndex   = threadStartIndex + N;
    int i;

    for( i=0; i<N; i++ ){
		C[threadStartIndex + (blockDim.x * i)] = A[threadStartIndex + (blockDim.x * i)] + B[threadStartIndex + (blockDim.x * i)];
    }
}
